#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define MASK_WIDTH 5
#define O_TILE_WIDTH 16
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE 
//implement the tiled 2D convolution kernel with adjustments for channels and make sure to:
//-use the constant memory for the convolution mask
//-use shared memory to reduce the number of global accesses and handle the boundary conditions when loading input list elements into the shared memory
//-clamp your output values

// P is output
// N is input
// M is cache
// Ns is shared memory
__global__ void convolutionKernal(float *P, float *N, int height, int width, int channels, const float * __restrict__ M) {
	__shared__ float Ns[O_TILE_WIDTH + 2 * MASK_WIDTH][O_TILE_WIDTH + 2 * MASK_WIDTH][3];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row_o = blockIdx.y * O_TILE_WIDTH + ty;
	int col_o = blockIdx.x * O_TILE_WIDTH + tx;

	int row_i = row_o - 2;
	int col_i = col_o - 2;

	// deal w/ boundaries
	for (int c = 0; c < channels; c++) {
		if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width)) {
			Ns[ty][tx][c] = N[(row_i * width + col_i) * channels + c];
		}
		else {
			Ns[ty][tx][c] = 0.0f;
		}
	}

	__syncthreads();

	// some threads do not calculate output
	if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {
		for (int c = 0; c < channels; c++) {
			float output = 0.0f;
			for (int i = 0; i < MASK_WIDTH; i++) {
				for (int j = 0; j < MASK_WIDTH; j++) {
					output += M[i * MASK_WIDTH + j] * Ns[i + ty][j + tx][c];
				}
			}

			if (row_o < height && col_o < width) {
				P[(row_o * width + col_o) * channels + c] = clamp(output);
			}
		}

	}

}

int main(int argc, char *argv[]) {
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(arg, 0);
  inputMaskFile  = wbArg_getInputFile(arg, 1);

  inputImage   = wbImport(inputImageFile);
  hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == MASK_WIDTH);    /* mask height is fixed to 5 */
  assert(maskColumns == MASK_WIDTH); /* mask width is fixed to 5 */

  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ INSERT CODE HERE
  int size = imageWidth * imageHeight * imageChannels * sizeof(float);
  hipMalloc((void**)&deviceInputImageData, size);
  hipMalloc((void**)&deviceOutputImageData, size);
  hipMalloc((void**)&deviceMaskData, (MASK_WIDTH * MASK_WIDTH * sizeof(float)));

  //allocate device memory
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(deviceInputImageData, hostInputImageData, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceOutputImageData, hostOutputImageData, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData, (MASK_WIDTH * MASK_WIDTH * sizeof(float)), hipMemcpyHostToDevice);

  //copy host memory to device
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 dimBlock(O_TILE_WIDTH + (MASK_WIDTH-1), O_TILE_WIDTH + (MASK_WIDTH-1));
  dim3 dimGrid((wbImage_getWidth(inputImage) - 1) / O_TILE_WIDTH + 1,
	  (wbImage_getHeight(inputImage) - 1) / O_TILE_WIDTH + 1, 1);

  convolutionKernal << <dimGrid, dimBlock >> > (deviceOutputImageData, deviceInputImageData, imageHeight, imageWidth, imageChannels, deviceMaskData);

  //initialize thread block and kernel grid dimensions
  //invoke CUDA kernel	
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(hostOutputImageData, deviceOutputImageData, size, hipMemcpyDeviceToHost);

  //copy results from device to host	
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(arg, outputImage);

  //@@ INSERT CODE HERE
  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  //deallocate device memory

  free(hostMaskData);
  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
